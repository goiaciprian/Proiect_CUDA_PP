#include "hip/hip_runtime.h"
#include "filters.cuh"

__global__ void filters::invertFilterKernel(utils::byte* original, utils::byte* newImg, int size) {
	int idThread = threadIdx.x, idBlock = blockIdx.x, id = idBlock * blockDim.x + idThread;

	unsigned int red = 255 - ((unsigned int)(original + id)[0]);
	unsigned int green = 255 - ((unsigned int)(original + id)[1]);
	unsigned int blue = 255 - ((unsigned int)(original + id)[2]);

	(newImg + id)[0] = (utils::byte)red;
	(newImg + id)[1] = (utils::byte)green;
	(newImg + id)[2] = (utils::byte)blue;
}


hipError_t filters::invertFilterRunner(int blocks, int threads, utils::byte* original, utils::byte* newImg, int size) {

	utils::byte* originalImg_d;
	utils::byte* newImage_d;

	hipMalloc((void**)&originalImg_d, size * sizeof(utils::byte));
	hipMalloc((void**)&newImage_d, size * sizeof(utils::byte));

	hipMemcpy(originalImg_d, original, size * sizeof(utils::byte), hipMemcpyHostToDevice);
	hipMemcpy(newImage_d, newImg, size * sizeof(utils::byte), hipMemcpyHostToDevice);

	filters::invertFilterKernel << <blocks, threads >> > (originalImg_d, newImage_d, size);

	hipMemcpy(newImg, newImage_d, size * sizeof(utils::byte), hipMemcpyDeviceToHost);

	hipFree(&originalImg_d);
	hipFree(&newImage_d);

	return hipGetLastError();

}
#include "hip/hip_runtime.h"
#include "filters.cuh"

__global__ void filters::sepiaFilterKernel(utils::byte* original, utils::byte* newImg, int size){
	int idThread = threadIdx.x, idBlock = blockIdx.x, id = idBlock * blockDim.x + idThread;

	unsigned int red = ((unsigned int)(original + id)[0]);
	unsigned int green = ((unsigned int)(original + id)[1]);
	unsigned int blue = ((unsigned int)(original + id)[2]);

	float tr = std::round( (0.393f * red) + (0.769f * green) + (0.189f * blue) );
	float tg = std::round( (0.349f * red) + (0.686f * green) + (0.168f * blue) );
	float tb = std::round( (0.272f * red) + (0.534f * green) + (0.131f * blue) ); 

	(newImg + id)[0] = (utils::byte)tr > 255 ? 255 : tr;
	(newImg + id)[1] = (utils::byte)tg > 255 ? 255 : tg;
	(newImg + id)[2] = (utils::byte)tb > 255 ? 255 : tb;
}

hipError_t filters::sepiaFilterRunner(int blocks, int threads, utils::byte* original, utils::byte* newImg, int size){
	utils::byte* original_d, *newImg_d;

	hipMalloc((void**)&original_d, size * sizeof(utils::byte));
	hipMalloc((void**)&newImg_d, size * sizeof(utils::byte));

	hipMemcpy(original_d, original, size * sizeof(utils::byte), hipMemcpyHostToDevice);
	hipMemcpy(newImg_d, newImg, size * sizeof(utils::byte), hipMemcpyHostToDevice);

	filters::sepiaFilterKernel <<<blocks, threads>>>(original_d, newImg_d, size);

	hipMemcpy(newImg, newImg_d, size * sizeof(utils::byte), hipMemcpyDeviceToHost);

	return hipGetLastError();
}
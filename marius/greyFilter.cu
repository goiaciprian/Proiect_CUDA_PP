#include "hip/hip_runtime.h"
#include "filters.cuh"


__global__ void filters::greyFilterKernel(utils::byte *image, utils::byte* newImage, int size) {
	int idThread = threadIdx.x, idBlock = blockIdx.x, id = idBlock * blockDim.x + idThread;

	unsigned int red = (unsigned int)(image + id)[0];
	unsigned int green = (unsigned int)(image + id)[1];
	unsigned int blue = (unsigned int)(image + id)[2];

	newImage[id] = (red + green + blue) / 3;
}


hipError_t filters::greyFilterRunner(int blocks, int threads, utils::byte* image, utils::byte* newImage, int size) {
	utils::byte* originalImage;

	utils::byte* newImage_d;

	hipMalloc((void**)&originalImage, size * sizeof(utils::byte));
	hipMalloc((void**)&newImage_d, size * sizeof(utils::byte));

	hipMemcpy(originalImage, image, size * sizeof(utils::byte), hipMemcpyHostToDevice);
	hipMemcpy(newImage_d, newImage, size * sizeof(utils::byte), hipMemcpyHostToDevice);

	filters::greyFilterKernel << < blocks, threads >> > (originalImage, newImage_d, size);

	hipMemcpy(newImage, newImage_d, size * sizeof(utils::byte), hipMemcpyDeviceToHost);

	hipFree(&originalImage);
	hipFree(&newImage_d);

	return hipGetLastError();
};

#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core.hpp>

#include <stdio.h>
#include <math.h>
#include <iostream>

__global__ void test2D(double* m1, double* m2, double* m3, int N)
{
    double PI = 3.14;
    int id_i = threadIdx.x + blockDim.x * blockIdx.x;
    int id_j = threadIdx.y + blockDim.y * blockIdx.y;

    if (id_i < N && id_j < N) {
        m1[id_i * N + id_j] = pow(sin((2 * PI * id_i) / N), 2) + pow(cos((2 * PI * id_j) / N), 2);
        m2[id_i * N + id_j] = pow(sin((2 * PI * id_i) / N), 2) + pow(cos((2 * PI * id_j) / N), 2);
    }
}

__global__ void test1D(double* m1, double* m2, double* m3, int N)
{
    double PI = 3.14;
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id < N * N) {
        int id_i = id / N;
        int id_j = id % N;
        m1[id_i * N + id_j] = pow(sin((2 * PI * id_i) / N), 2) + pow(cos((2 * PI * id_j) / N), 2);
        m2[id_i * N + id_j] = pow(sin((2 * PI * id_i) / N), 2) + pow(cos((2 * PI * id_j) / N), 2);
    }
}

__global__ void addMatrix1D(double* m1, double* m2, double* m3, int N) {
    int thread = threadIdx.x;
    int block = blockIdx.x;

    int id = block * blockDim.x + thread;
    if (id < N * N) m3[id] = m1[id] + m2[id];
}

int main()
{
    int N = 1024;

    double* m1_h = new double[N * N];
    double* m2_h = new double[N * N];
    double* m3_h = new double[N * N];

    double* m1_d;
    double* m2_d;
    double* m3_d;

    hipMalloc((void**)&m1_d, N * N * sizeof(double));
    hipMalloc((void**)&m2_d, N * N * sizeof(double));
    hipMalloc((void**)&m3_d, N * N * sizeof(double));

    hipMemcpy(m1_d, m1_h, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(m2_d, m2_h, N * N * sizeof(double), hipMemcpyHostToDevice);
    

    dim3 threadsPerBlock(16, 32);
    dim3 blocks2D(N / 16, N / 32);
    dim3 blocks1D(N * N / 512 + 1, 512); // 512 = 16*32

    test1D << <blocks1D, threadsPerBlock >> > (m1_d, m2_d, m3_d, N);

    addMatrix1D << <blocks1D, threadsPerBlock >> > (m1_d, m2_d, m3_d, N);

    hipMemcpy(m3_h, m3_d, N * N * sizeof(double), hipMemcpyDeviceToHost);


    for (int i = 0; i < N; i++)
    {
        /*
        std::cout<<m3_h[0]<<std::endl;
        std::cout << m3_h[1] << std::endl;
        */
        std::cout << m3_h[i] << std::endl;
    }
}
#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core.hpp>

#include <stdio.h>
#include <math.h>
#include <iostream>

#include "utils.h"

#define BLOCK_SIZE 16.0

__global__ void sobelFilter(unsigned char* original, unsigned char* filterResult, const unsigned int width, const unsigned int height) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	float gx = 0, gy = 0;

	//printf("%d\n", (int)original[y * width + x]);
	//if ((x > 0 && x < width - 1) || (y > 0 && y < height - 1)) {
	if (x > 0 && x < width * height - 1) {

		/*gx = (-1 * original[(y - 1) * width + (x - 1)]) + 
				(-2 * original[y * width + (x - 1)]) + 
				(-1 * original[(y + 1) * width + (x - 1)]) +
				(original[(y - 1) * width + (x + 1)]) + 
				(2 * original[y * width + (x + 1)]) + 
				(original[(y + 1) * width + (x + 1)]);
		
		gy = (original[(y - 1) * width + (x - 1)]) + 
				(2 * original[(y - 1) * width + x]) + 
				(original[(y - 1) * width + (x + 1)]) +
				(-1 * original[(y + 1) * width + (x - 1)]) + 
				(-2 * original[(y + 1) * width + x]) + 
				(-1 * original[(y + 1) * width + (x + 1)]);*/

		gx = (-1 * original[(x - 1)]) +
			(-2 * original[(x - 1)]) +
			(-1 * original[(x - 1)]) +
			(original[(x + 1)]) +
			(2 * original[(x + 1)]) +
			(original[(x + 1)]);

		gy = (original[(x - 1)]) +
			(2 * original[x]) +
			(original[(x + 1)]) +
			(-1 * original[(x - 1)]) +
			(-2 * original[x]) +
			(-1 * original[(x + 1)]);

		//printf("%d\n", (int) original[y*width + x]);

		//todo aici e eroare
		filterResult[y * width + x] = (unsigned char) sqrt(gx * gx + gy * gy);
		
		//printf("%d %d %d\n", blockDim.x, blockIdx.x, threadIdx.x);
		
		//printf("%d\n",(blockDim.x + blockIdx.x) + threadIdx.x);


		//filterResult[y * width + x] = gx;
	
	}
}

__global__ void initImgArr(unsigned char* image, const unsigned int width, const unsigned int height) {
	int x = threadIdx.x + blockIdx.x + blockDim.x;
	int y = threadIdx.y + (blockIdx.y * blockDim.y);



	//if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		image[y * width + x] = 0;
		//printf("hello from init %d\n", x);

	//}
}

__global__ void pixelsCopy(const unsigned char* original, unsigned char* filterResult, const unsigned int width, const unsigned int height) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);

	/*if (x >= 0 && x < width && y >= 0 && y < height) {
	}*/
		filterResult[y * width + x] = (unsigned char)(original[y * width + x]);
}

__global__ void aduna(int* a, int* b, int c) {
	int i = threadIdx.x;
	printf("%d", a[i] + b[i]);
}

int main()
{
	std::string filePath = cv::samples::findFile("sobel.png");

	cv::Mat img = cv::imread(filePath, cv::IMREAD_COLOR);
	cv::Mat imgGray = cv::imread(filePath, cv::IMREAD_GRAYSCALE);

	if (img.empty()) {
		return 1;
	}

	unsigned char* imgArr = utils::matToBytes(imgGray), * imgArr_d;

	hipMalloc(&imgArr_d, img.total() * img.elemSize() * sizeof(unsigned char));
	hipMemcpy(imgArr_d, imgArr, img.total() * img.elemSize() * sizeof(unsigned char), hipMemcpyHostToDevice);

	//---CUDA START---
	unsigned char* filterResult_h = new unsigned char[imgGray.total() * imgGray.elemSize()]; //Host

	unsigned char* filterResult_d; //Device

	hipMalloc((void**)&filterResult_d, imgGray.total() * imgGray.elemSize() * sizeof(unsigned char));

	hipMemcpy(filterResult_d, filterResult_h, imgGray.total() * imgGray.elemSize() * sizeof(unsigned char), hipMemcpyHostToDevice);

	//hipMemset(filterResult_d, 0, imgGray.total() * imgGray.elemSize());

	//Blocks & threads
	dim3 threadsPerBlock((int)BLOCK_SIZE, (int)BLOCK_SIZE);
	dim3 blocks((int)ceil(imgGray.rows / (int)BLOCK_SIZE), (int)ceil(imgGray.cols / (int)BLOCK_SIZE));

	dim3 threads2(img.total() * img.elemSize() * sizeof(unsigned char));

	//initImgArr << < ceil(img.total() * img.elemSize()) / 1024, 1024 >> > (filterResult_d, imgGray.cols, imgGray.rows);
	sobelFilter << < ceil(img.total() * img.elemSize() / 1024), 1024>> > (imgArr_d, filterResult_d, img.cols, img.rows);
	//pixelsCopy << <ceil(img.total() * img.elemSize() / 1024), 1024 >> > (imgArr_d, filterResult_d, img.cols, img.rows);

	/*int* a = new int[2], * b = new int[2];
	a[0] = 2;
	a[1] = 4;

	b[0] = 6;
	b[1] = 12;

	int* a_d, * b_d;

	hipMalloc(&a_d, 2 * sizeof(int));
	hipMalloc(&b_d, 2 * sizeof(int));

	hipMemcpy(a_d, a, 2* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, 2* sizeof(int), hipMemcpyHostToDevice);

	aduna << <1, 2 >> > (a_d, b_d, 0);*/

	hipError_t error = hipGetLastError();

	if (error != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(error) << std::endl;
	}

	hipDeviceSynchronize();

	hipMemcpy(filterResult_h, filterResult_d, imgGray.total() * imgGray.elemSize() * sizeof(unsigned char), hipMemcpyDeviceToHost);
	//---CUDA END---
	

	//cv::Mat mat = utils::bytesToMat(imgArr, img.cols, img.rows);
	cv::Mat sobel = utils::bytesToMat(filterResult_h, imgGray.cols, imgGray.rows);

	cv::imwrite("test.jpeg", sobel);

	cv::imshow("original", img);
	cv::imshow("originalGray", imgGray);
	cv::imshow("sobel", sobel);

	cv::waitKey(0);

	// Dimensiune imagine
	std::cout << "rows: " << imgGray.rows << " cols: " << imgGray.cols << " total: " << imgGray.total() << " elemSize: " << imgGray.elemSize() << " step[0]: " << imgGray.step[0];

	// Asa se iau valorile de RGB dintr-un pixel
	/*int x = 50, y = 30; 
	std::cout << (int)img.at<cv::Vec3b>(y, x)[0] << std::endl;
	std::cout << (int)img.at<cv::Vec3b>(y, x)[1] << std::endl;
	std::cout << (int)img.at<cv::Vec3b>(y, x)[2] << std::endl;
	*/

	//WTF
	//for (int i = 0; i < img.total() * img.elemSize(); i++) std::cout << (int)imgArr[i] + " ";

	//for (int i = 0; i < img.total() * img.elemSize(); i++) std::cout << (int)(filterResult_h+i)<<" ";

	hipFree(&filterResult_d);
	hipFree(&imgArr);

	return 0;
}
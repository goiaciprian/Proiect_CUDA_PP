#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <stdio.h>
#include <iostream>

#include "utils.h"
#include "filters.cuh"


int main()
{
	std::string filePath = cv::samples::findFile("sobel.png");

	cv::Mat img = cv::imread(filePath, cv::IMREAD_COLOR);
	cv::Mat imgGreyscale = cv::imread(filePath, cv::IMREAD_GRAYSCALE);

	int size = img.total() * img.elemSize();
	int threads = 1024;
	int blocks = ceil(size / threads);

	utils::byte* forSobel = utils::matToBytes(imgGreyscale);

	utils::byte *imagine = utils::matToBytes(img);
	utils::byte *newImage_grey = new utils::byte[size];
	utils::byte* newImage_inverted = new utils::byte[size];
	utils::byte* newImage_sobel = new utils::byte[size];

	hipError_t status_grey = filters::greyFilterRunner(blocks, threads, imagine, newImage_grey, size);

	if (status_grey != hipSuccess) {
		std::cout << "Eroare cuda: " << hipGetErrorString(status_grey) << std::endl;
	}
	hipError_t status_invert = filters::invertFilterRunner(blocks, threads, imagine, newImage_inverted, size);
	
	if (status_invert != hipSuccess) {
		std::cout << "Eroare cuda: " << hipGetErrorString(status_invert) << std::endl;
	}

	hipError_t status_sobel = filters::sobelFilterRunner(blocks, threads, forSobel, newImage_sobel, size, img.rows, img.cols);

	if (status_sobel != hipSuccess) {
		std::cout << "Eroare cuda: " << hipGetErrorString(status_sobel) << std::endl;
	}

	std::cout << CV_8U;

	cv::Mat newimg_grey = utils::bytesToMat(newImage_grey, img.cols, img.rows, CV_8UC3);
	cv::Mat newimg_invert = utils::bytesToMat(newImage_inverted, img.cols, img.rows, CV_8UC3);
	cv::Mat newimg_sobel = utils::bytesToMat(newImage_sobel, img.cols, img.rows, CV_8U);


	cv::imshow("Original", img);
	cv::imshow("Greyfilter", newimg_grey);
	cv::imshow("Inverted", newimg_invert);
	cv::imshow("Sobel", newimg_sobel);

	int test = cv::waitKey();


	return 0;
}
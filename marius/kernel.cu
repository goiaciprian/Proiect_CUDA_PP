﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core.hpp>

#include <stdio.h>
#include <math.h>
#include <iostream>

#include "utils.h"

int main()
{
	std::string filePath = cv::samples::findFile("thisisdog.jpeg");
	cv::Mat img = cv::imread(filePath, cv::IMREAD_COLOR);

	if (img.empty()) {
		return 1;
	}

	uint8_t* imgArr = utils::matToBytes(img);

	cv::Mat mat = utils::bytesToMat(imgArr, img.cols, img.rows);

	cv::imwrite("test.jpeg", mat);
   
}
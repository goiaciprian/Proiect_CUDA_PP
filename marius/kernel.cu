#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <stdio.h>
#include <iostream>

#include "utils.h"
#include "filters.cuh"

int main()
{
	std::string filePath = cv::samples::findFile("thisisdog.jpeg");
	cv::Mat img = cv::imread(filePath, cv::IMREAD_COLOR);

	int size = img.total() * img.elemSize();
	int threads = 1024;
	int blocks = ceil(size / threads);

	utils::byte *imagine = utils::matToBytes(img);
	utils::byte *newImage_grey = new utils::byte[size];
	utils::byte* newImage_inverted = new utils::byte[size];


	hipError_t status = filters::greyFilterRunner(blocks, threads, imagine, newImage_grey, size);

	if (status != hipSuccess) {
		std::cout << "Eroare cuda: " << hipGetErrorString(status) << std::endl;
	}

	hipError_t status_invert = filters::invertFilterRunner(blocks, threads, imagine, newImage_inverted, size);
	
	if (status_invert != hipSuccess) {
		std::cout << "Eroare cuda: " << hipGetErrorString(status) << std::endl;
	}


	cv::Mat newimg_grey = utils::bytesToMat(newImage_grey, img.cols, img.rows);
	cv::Mat newimg_invert = utils::bytesToMat(newImage_inverted, img.cols, img.rows);


	cv::imshow("Original", img);
	cv::imshow("Greyfilter", newimg_grey);
	cv::imshow("Inverted", newimg_invert);

	int test = cv::waitKey();


	return 0;
}
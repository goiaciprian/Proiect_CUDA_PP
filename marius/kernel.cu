#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <stdio.h>
#include <iostream>

#include "utils.h"
#include "filters.cuh"


int main()
{
	while(1) {
		std::string fileName = "";
		int input = 0;

		std::cout << "Alegeti poza:\n1.Bucsa\n2.Hippo \n3.Base\n4.Sobel\n5.Creation\n6.Starea noastra mentala\n7.NFT \n\nAlegere: ";
		std::cin >> input;

		switch(input) {
		case 1:
			fileName = "bucsa.jpg";
			break;
		case 2:
			fileName = "jordan.jpeg";
			break;
		case 3:
			fileName = "base.jpg";
			break;
		case 4:
			fileName = "sobel.png";
			break;
		case 5:
			fileName = "creation.jpeg";
			break;
		case 6:
			fileName = "starementala.jpeg";
			break;
		case 7:
			fileName = "nft.jpeg";
			break;

		default:
			return;
		}

		system("cls");

		std::string filePath = cv::samples::findFile(fileName);

		cv::Mat img = cv::imread(filePath, cv::IMREAD_COLOR);
	
		int size = img.total() * img.elemSize();
		int threads = 1024;
		int blocks = ceil(size / threads);

		utils::byte *imagine = utils::matToBytes(img);

		utils::byte *newImage_grey = new utils::byte[size];
		utils::byte* newImage_inverted = new utils::byte[size];
		utils::byte* newImage_sobel = new utils::byte[img.total()];
		utils::byte* newImage_sepia = new utils::byte[size];

		hipError_t status_grey = filters::greyFilterRunner(blocks, threads, imagine, newImage_grey, size);

		if (status_grey != hipSuccess) {
			std::cout << "Eroare cuda grey : " << hipGetErrorString(status_grey) << std::endl;
		}
		hipError_t status_invert = filters::invertFilterRunner(blocks, threads, imagine, newImage_inverted, size);
	
		if (status_invert != hipSuccess) {
			std::cout << "Eroare cuda invert: " << hipGetErrorString(status_invert) << std::endl;
		}

		utils::byte* grey1Channel = utils::ch3toCh1(newImage_grey, img.total(), size);


		hipError_t status_sobel = filters::sobelFilterRunner(blocks, threads, grey1Channel, newImage_sobel, img.total(), img.rows, img.cols);

		if (status_sobel != hipSuccess) {
			std::cout << "Eroare cuda: " << hipGetErrorString(status_sobel) << std::endl;
		}

		hipError_t status_sepia = filters::sepiaFilterRunner(blocks, threads, imagine, newImage_sepia, size);
		if (status_sepia != hipSuccess) {
			std::cout << "Eroare cuda: " << hipGetErrorString(status_sobel) << std::endl;
		}


		cv::Mat newimg_grey = utils::bytesToMat(grey1Channel, img.cols, img.rows, CV_8U);
		cv::Mat newimg_invert = utils::bytesToMat(newImage_inverted, img.cols, img.rows, CV_8UC3);
		cv::Mat newimg_sobel = utils::bytesToMat(newImage_sobel, img.cols, img.rows, CV_8U);
		cv::Mat newimg_sepia = utils::bytesToMat(newImage_sepia, img.cols, img.rows, CV_8UC3);

		cv::imshow("Original", img);
		cv::imshow("Greyfilter", newimg_grey);
		cv::imshow("Inverted", newimg_invert);
		cv::imshow("Sobel", newimg_sobel);
		cv::imshow("Drog", newimg_sepia);


		int test = cv::waitKey();
		cv::destroyAllWindows();
		system("cls");
	}

	return 0;
}
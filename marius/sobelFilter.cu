#include "hip/hip_runtime.h"
#include "filters.cuh"


__global__ void filters::sobelFilterKernel(utils::byte* original, utils::byte* filterResult, int , unsigned int width, unsigned int height) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	float gx = 0, gy = 0;

	if (x > 0 && x < width * height - 1) {


		gx = (-1 * original[(x - 1)]) +
			(-2 * original[(x - 1)]) +
			(-1 * original[(x - 1)]) +
			(original[(x + 1)]) +
			(2 * original[(x + 1)]) +
			(original[(x + 1)]);

		gy = (original[(x - 1)]) +
			(2 * original[x]) +
			(original[(x + 1)]) +
			(-1 * original[(x - 1)]) +
			(-2 * original[x]) +
			(-1 * original[(x + 1)]);

		filterResult[y * width + x] = (utils::byte)sqrt(gx * gx + gy * gy);
	}
}


hipError_t filters::sobelFilterRunner(int blocks, int threads, utils::byte* original_h, utils::byte* filterResult_h, int size, unsigned int width, unsigned int height) {
	utils::byte* original_d, *filterResult_d;

	hipMalloc((void**) & original_d, size * sizeof(utils::byte));
	hipMalloc((void**) & filterResult_d, size * sizeof(utils::byte));

	hipMemcpy(original_d, original_h, size * sizeof(utils::byte), hipMemcpyHostToDevice);
	hipMemcpy(filterResult_d, filterResult_h, size * sizeof(utils::byte), hipMemcpyHostToDevice);

	filters::sobelFilterKernel << <blocks, threads >> > (original_d, filterResult_d, size, width, height);

	hipMemcpy(filterResult_h, filterResult_d, size * sizeof(utils::byte), hipMemcpyDeviceToHost);

	hipFree(&original_d);
	hipFree(&filterResult_d);


	return hipGetLastError();
}